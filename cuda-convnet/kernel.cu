#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "data_descriptor.cuh"

#define threadsPerBlock 1024
__constant__ int N;


__global__ void dot(float *a,
	float *b, float *c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float   temp = 0;
	while (tid < N) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	// set the cache values
	cache[cacheIndex] = temp;

	// synchronize threads in this block
	__syncthreads();

	// for reductions, threadsPerBlock must be a power of 2
	// because of the following code
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		atomicAdd(c, cache[0]);
	}
}

int main()
{
	int h_N = 10000;
	float *h_a, *h_b, *h_z;
	float *d_a, *d_b, *d_z;

	h_a = (float*)malloc(sizeof(float)*h_N);
	h_b = (float*)malloc(sizeof(float)*h_N);
	h_z = (float*)malloc(sizeof(float)*h_N);
	hipMalloc((void**)&d_a, sizeof(float)*h_N);
	hipMalloc((void**)&d_b, sizeof(float)*h_N);
	hipMalloc((void**)&d_z, sizeof(float)*h_N);
	hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(h_N));

	for (int i = 0; i < h_N; i++){
		h_a[i] = 1;
		h_b[i] = 1;
	}

	hipMemcpy(d_a, h_a, sizeof(float) * h_N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(float) * h_N, hipMemcpyHostToDevice);
	int num_block = h_N % threadsPerBlock ? h_N / threadsPerBlock + 1 : h_N / threadsPerBlock;
	dot <<< num_block, threadsPerBlock >>>(d_a, d_b, d_z);
	hipDeviceSynchronize();
	hipMemcpy(h_z, d_z, sizeof(float) * h_N, hipMemcpyDeviceToHost);
	printf("%f\n", *h_z);
	getchar();
    return 0;
}
