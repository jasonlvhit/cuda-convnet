#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "data_descriptor.cuh"

#define threadsPerBlock 1024
__constant__ int out_area, in_area;

__device__ void dot(const float *a,
	const float *b, float *c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float   temp = 0;
	while (tid < in_area) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	// set the cache values
	cache[cacheIndex] = temp;

	// synchronize threads in this block
	__syncthreads();

	// for reductions, threadsPerBlock must be a power of 2
	// because of the following code
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		atomicAdd(c, cache[0]);
	}
}

/*
Weight format:
===============
| (x0, y0) | (x1, y0) | (x2, y0) | (x3, y0) |... |xN, y0) |
| (x1, y0) |...
...
| (xN, y0)

*/

__global__ void cnnFullyConnectedLayerForward(const float *input,
	const float *W, const float *b, float *output)
{
	for (int i = 0; i < out_area; i++){
		dot(input, W + in_area * i, output + i);
		output[i] = 1.0 / (1.0 + expf(-(output[i] + b[i])));
	}
}

int main()
{
	/*
	int h_N = 10000;
	float *h_a, *h_b, *h_z;
	float *d_a, *d_b, *d_z;

	h_a = (float*)malloc(sizeof(float)*h_N);
	h_b = (float*)malloc(sizeof(float)*h_N);
	h_z = (float*)malloc(sizeof(float)*h_N);
	hipMalloc((void**)&d_a, sizeof(float)*h_N);
	hipMalloc((void**)&d_b, sizeof(float)*h_N);
	hipMalloc((void**)&d_z, sizeof(float)*h_N);
	hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(h_N));

	
	for (int i = 0; i < h_N; i++){
		h_a[i] = 1;
		h_b[i] = 1;
	}

	hipMemcpy(d_a, h_a, sizeof(float) * h_N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(float) * h_N, hipMemcpyHostToDevice);
	int num_block = h_N % threadsPerBlock ? h_N / threadsPerBlock + 1 : h_N / threadsPerBlock;
	dot <<< num_block, threadsPerBlock >>>(d_a, d_b, d_z);
	hipDeviceSynchronize();
	hipMemcpy(h_z, d_z, sizeof(float) * h_N, hipMemcpyDeviceToHost);
	printf("%f\n", *h_z);
	*/
	float *h_input, *h_output, *h_W, *h_b;
	float *d_input, *d_output, *d_W, *d_b;
	int h_in_area = 10000;
	int h_out_area = 10000;
	int d_in_area;
	int d_out_area;

	h_input = (float*)malloc(sizeof(float)*h_in_area);
	h_output = (float*)malloc(sizeof(float)*h_out_area);
	h_W = (float*)malloc(sizeof(float)*h_in_area *h_out_area);
	h_b = (float*)malloc(sizeof(float)*h_out_area);
	hipMalloc((void**)&d_input, sizeof(float)*h_in_area);
	hipMalloc((void**)&d_output, sizeof(float)*h_out_area);
	hipMalloc((void**)&d_W, sizeof(float)*h_in_area *h_out_area);
	hipMalloc((void**)&d_b, sizeof(float)*h_out_area);
	hipMalloc((void**)&d_in_area, sizeof(int));
	hipMalloc((void**)&d_out_area, sizeof(int));

	for (int i = 0; i < h_in_area; i++){
		h_input[i] = 1;
	}

	for (int i = 0; i < h_in_area * h_out_area; i++){
		h_W[i] = 1;
	}

	for (int i = 0; i < h_out_area; i++){
		h_output[i] = 0;
		h_b[i] = 1;
	}

	hipMemcpy(d_input, h_input, sizeof(float)*h_in_area, hipMemcpyHostToDevice);
	hipMemcpy(d_W, h_W, sizeof(float)*h_in_area*h_out_area, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(float)*h_out_area, hipMemcpyHostToDevice);
	hipMemcpy(&d_in_area, &h_in_area, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_output, h_output, sizeof(float)*h_out_area, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(in_area), &h_in_area, sizeof(h_in_area));
	hipMemcpyToSymbol(HIP_SYMBOL(out_area), &h_out_area, sizeof(h_out_area));
	int num_block = h_out_area % threadsPerBlock ? h_out_area / threadsPerBlock + 1 : h_out_area / threadsPerBlock;
	cnnFullyConnectedLayerForward << < num_block, threadsPerBlock >> >(d_input, d_W, d_b, d_output);
	hipDeviceSynchronize();
	hipMemcpy(h_output, d_output, sizeof(float)*h_out_area, hipMemcpyDeviceToHost);

	for (int i = 0; i < h_out_area/10; i++){
		printf("%f\t", h_output[i]);
	}

	getchar();
    return 0;
}
