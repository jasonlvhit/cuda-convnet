#include "hip/hip_runtime.h"

#include <vector>

#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "data_descriptor.cuh"
#include "mnist_parser.cuh"
using namespace std;

#define KERNEL_WIDTH 5
#define threadsPerBlock 1024
__constant__ int out_area, in_area;

__constant__ int kernel_width, in_width, in_height, out_width;

__global__ void convolution(const float* input, const float* W, float * output){
	int out_id = blockIdx.x + out_width * blockIdx.y;
	int cache_id = threadIdx.x + threadIdx.y * kernel_width;
	__shared__ float cache[32];
	if (out_id < out_area){
		cache[cache_id] = W[cache_id] *
			input[(blockIdx.y + threadIdx.y) * in_width + blockIdx.x + threadIdx.x];
	}
	for (int i = 25; i < 32; i++) cache[i] = 0;
	__syncthreads();

	int i =  16;
	cache_id = threadIdx.x + threadIdx.y * blockDim.x;
	while (i != 0) {
		if (cache_id < i)
			cache[cache_id] += cache[cache_id + i];
		__syncthreads();
		i /= 2;
	}
	if (cache_id == 0 && out_id < out_area) {	
		output[out_id] += cache[0];
	}
}


void train()
{
	vector<vector<float> > train_x;
	vector<float> train_y;
	LOAD_MNIST_TRAIN(train_x, train_y);

	float x[60000][28 * 28];
	float y[60000];

	for (int i = 0; i < train_x.size(); i++){
		for (int j = 0; j < train_x[i].size(); j++){
			x[i][j] = train_x[i][j];
		}
		y[i] = train_y[i];
	}

	float **d_x, *d_y;
	
	/*cnnConvolutionalLayerDataDescriptor *C1 = new cnnConvolutionalLayerDataDescriptor(28, 28, 1, 6, 5, 5);
	cnnMaxPoolingLayerDataDescriptor *S2 = new cnnMaxPoolingLayerDataDescriptor(24, 24, 6);
	cnnConvolutionalLayerDataDescriptor *C3 = new cnnConvolutionalLayerDataDescriptor(12, 12, 6, 100, 12, 12);
	cnnFullyConnectedLayerDataDescriptor *F4 = new cnnFullyConnectedLayerDataDescriptor(100, 10);*/



	hipMalloc((void**)d_x, sizeof(float) * 60000 * 28 * 28);
	hipMalloc((void**)d_y, sizeof(float) * 60000);

	float* d_C1_input, *d_C1_output, *d_C1_W, *d_C1_b, *d_C1_err_terms;
	float *d_S2_input, *d_S2_output, *d_S2_err_terms;
	float *d_C3_input, *d_C3_output, *d_C3_W, *d_C3_b, *d_C3_err_terms;
	float *d_F4_input, *d_F4_output, *d_F4_W, *d_F4_b, *d_F4_err_terms;
	float *d_O_input, *d_O_err_terms;

	hipMalloc((void**)d_C1_input, sizeof(float) * 28 * 28);
	hipMalloc((void**)d_C1_output, sizeof(float) * 24 * 24 * 6);
	hipMalloc((void**)d_C1_W, sizeof(float) * 5 * 5 * 6);
	hipMalloc((void**)d_C1_b, sizeof(float) * 24 * 24 * 6);
	hipMalloc((void**)d_C1_err_terms, sizeof(float) * 28 * 28);

	hipMalloc((void**)d_S2_input, sizeof(float) * 24 * 24 * 6);
	hipMalloc((void**)d_S2_output, sizeof(float) * 12 * 12 * 6);
	hipMalloc((void**)d_S2_err_terms, sizeof(float) * 24 * 24 * 6);

	hipMalloc((void**)d_C3_input, sizeof(float) * 12 * 12 * 6);
	hipMalloc((void**)d_C3_output, sizeof(float) * 1 * 1 * 100);
	hipMalloc((void**)d_C3_W, sizeof(float) * 12 * 12 * 6 * 100);
	hipMalloc((void**)d_C3_b, sizeof(float)* 1 * 1 * 100);
	hipMalloc((void**)d_C3_err_terms, sizeof(float) * 12 * 12 * 6);
	
	hipMalloc((void**)d_F4_input, sizeof(float) * 100);
	hipMalloc((void**)d_F4_output, sizeof(float) * 10);
	hipMalloc((void**)d_F4_W, sizeof(float) * 100 * 10);
	hipMalloc((void**)d_F4_b, sizeof(float) * 10);
	hipMalloc((void**)d_F4_err_terms, sizeof(float) * 100);

	hipMalloc((void**)d_O_err_terms, sizeof(float) * 10);
	hipMalloc((void**)d_O_input, sizeof(float) * 10);

	hipMemcpy(d_x, x, sizeof(float) * 60000 * 28 * 28, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(float) * 60000, hipMemcpyHostToDevice);

	

}

int main()
{
	/*
	int h_N = 10000;
	float *h_a, *h_b, *h_z;
	float *d_a, *d_b, *d_z;

	h_a = (float*)malloc(sizeof(float)*h_N);
	h_b = (float*)malloc(sizeof(float)*h_N);
	h_z = (float*)malloc(sizeof(float)*h_N);
	hipMalloc((void**)&d_a, sizeof(float)*h_N);
	hipMalloc((void**)&d_b, sizeof(float)*h_N);
	hipMalloc((void**)&d_z, sizeof(float)*h_N);
	hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(h_N));

	
	for (int i = 0; i < h_N; i++){
		h_a[i] = 1;
		h_b[i] = 1;
	}

	hipMemcpy(d_a, h_a, sizeof(float) * h_N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(float) * h_N, hipMemcpyHostToDevice);
	int num_block = h_N % threadsPerBlock ? h_N / threadsPerBlock + 1 : h_N / threadsPerBlock;
	dot <<< num_block, threadsPerBlock >>>(d_a, d_b, d_z);
	hipDeviceSynchronize();
	hipMemcpy(h_z, d_z, sizeof(float) * h_N, hipMemcpyDeviceToHost);
	printf("%f\n", *h_z);
	
	float *h_input, *h_output, *h_W, *h_b;
	float *d_input, *d_output, *d_W, *d_b;
	int h_in_area = 1000;
	int h_out_area = 1024;
	int d_in_area;
	int d_out_area;

	h_input = (float*)malloc(sizeof(float)*h_in_area);
	h_output = (float*)malloc(sizeof(float)*h_out_area);
	h_W = (float*)malloc(sizeof(float)*h_in_area *h_out_area);
	h_b = (float*)malloc(sizeof(float)*h_out_area);
	hipMalloc((void**)&d_input, sizeof(float)*h_in_area);
	hipMalloc((void**)&d_output, sizeof(float)*h_out_area);
	hipMalloc((void**)&d_W, sizeof(float)*h_in_area *h_out_area);
	hipMalloc((void**)&d_b, sizeof(float)*h_out_area);
	hipMalloc((void**)&d_in_area, sizeof(int));
	hipMalloc((void**)&d_out_area, sizeof(int));

	for (int i = 0; i < h_in_area; i++){
		h_input[i] = 1;
	}

	for (int i = 0; i < h_in_area * h_out_area; i++){
		h_W[i] = 1;
	}

	for (int i = 0; i < h_out_area; i++){
		h_output[i] = 0;
		h_b[i] = 1;
	}

	hipMemcpy(d_input, h_input, sizeof(float)*h_in_area, hipMemcpyHostToDevice);
	hipMemcpy(d_W, h_W, sizeof(float)*h_in_area*h_out_area, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(float)*h_out_area, hipMemcpyHostToDevice);
	hipMemcpy(&d_in_area, &h_in_area, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_output, h_output, sizeof(float)*h_out_area, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(in_area), &h_in_area, sizeof(h_in_area));
	hipMemcpyToSymbol(HIP_SYMBOL(out_area), &h_out_area, sizeof(h_out_area));
	int num_block = h_out_area % threadsPerBlock ? h_out_area / threadsPerBlock + 1 : h_out_area / threadsPerBlock;
	cnnFullyConnectedLayerForward << < h_out_area, threadsPerBlock >> >(d_input, d_W, d_b, d_output);
	hipDeviceSynchronize();
	hipMemcpy(h_output, d_output, sizeof(float)*h_out_area, hipMemcpyDeviceToHost);

	for (int i = 0; i < h_out_area; i++){
		printf("%f\t", h_output[i]);
	}
	*/

	float* h_input, *h_output, *h_W;
	int h_in_width = 16;
	int h_in_height = 16;
	int h_in_area = h_in_height * h_in_width;
	int h_kernel_width = 5;
	int h_kernel_height = 5;
	int h_out_width = h_in_width - h_kernel_width + 1;
	int h_out_height = h_in_height - h_kernel_height + 1;
	int h_out_area = h_out_width * h_out_height;
	h_input = (float*)malloc(sizeof(float)*h_in_area);
	h_output = (float*)malloc(sizeof(float)*h_out_area);
	h_W = (float*)malloc(sizeof(float)*h_kernel_width * h_kernel_height);

	for (int i = 0; i < h_in_area; i++){
		h_input[i] = 1.0;
	}

	for (int i = 0; i < h_out_area; i++){
		h_output[i] = 0.0;
	}

	for (int i = 0; i < h_kernel_height * h_kernel_width; i++){
		h_W[i] = 1.0;
	}

	float* d_input, *d_output, *d_W;
	hipMalloc((void**)&d_input, sizeof(float)*h_in_area);
	hipMalloc((void**)&d_output, sizeof(float)*h_out_area);
	hipMalloc((void**)&d_W, sizeof(float)*h_kernel_width * h_kernel_height);
	hipMemcpy(d_input, h_input, sizeof(float)*h_in_area, hipMemcpyHostToDevice);
	hipMemcpy(d_W, h_W, sizeof(float)*h_kernel_width*h_kernel_width, hipMemcpyHostToDevice);
	hipMemcpy(d_output, h_output, sizeof(float)*h_out_area, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(in_area), &h_in_area, sizeof(h_in_area));
	hipMemcpyToSymbol(HIP_SYMBOL(out_area), &h_out_area, sizeof(h_out_area));
	hipMemcpyToSymbol(HIP_SYMBOL(in_width), &h_in_width, sizeof(h_in_width));
	hipMemcpyToSymbol(HIP_SYMBOL(out_width), &h_out_width, sizeof(h_out_width));
	hipMemcpyToSymbol(HIP_SYMBOL(kernel_width), &h_kernel_width, sizeof(h_kernel_height));
	hipMemcpyToSymbol(HIP_SYMBOL(in_height), &h_in_height, sizeof(h_in_height));
	dim3 grid(h_out_width, h_out_height);
	dim3 block(8, 8);
	convolution<<< grid, block >>>(d_input, d_W, d_output);
	hipDeviceSynchronize();
	hipMemcpy(h_output, d_output, sizeof(float)*h_out_area, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < h_out_area; i++){
		printf("%f\t", h_output[i]);
	}
	
	
	getchar();
    return 0;
}
